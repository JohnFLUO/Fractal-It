#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Color/ColorSmooth/ColorSmooth.hpp"
#include <stdio.h>

#include <hip/hip_vector_types.h>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <math_functions.h>

__device__ unsigned short process(double2 startValue, unsigned short max_iters)  {

  double2 startZ = startValue;

  double2 z1;
  z1.x = startZ.x;
  z1.y = startZ.y;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    double2 z2;
    z2.x = z1.x * z1.x;
    z2.y = z1.y * z1.y;

    z1.y = 2.0f * z1.x * z1.y + startZ.y;
    z1.x = z2.x - z2.y + startZ.x;
    if ( (z2.x + z2.y) > 4.0f) {
      return counter;
    }
  }

  return max_iters - 1;
}

__global__ void kernel_compute(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH + x] = process(z, max_iters);
	}
}



void compute (int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}
