#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Color/ColorSmooth/ColorSmooth.hpp"
#include <stdio.h>
//#include "Convergence/double/Convergence_dp_x86.hpp"


__device__ unsigned int process(const double startReal, const double startImag, unsigned int max_iters)  {
  double zReal = startReal;
  double zImag = startImag;

  for (unsigned int counter = 0; counter < max_iters; counter++) {
    double r2 = zReal * zReal;
    double i2 = zImag * zImag;
    zImag = 2.0f * zReal * zImag + startImag;
    zReal = r2 - i2 + startReal;
    if ( (r2 + i2) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}


__global__ void kernel_saxpy(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, double *device_value) {
	int y = blockIdx.x * blockDim.x + threadIdx.x;
  double imag;
  double real;

	if ( y < IMAGE_HEIGHT ) {
    imag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    for (int x = 0; x < IMAGE_WIDTH;  x++) {
      real = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
      device_value[y*IMAGE_WIDTH +x] = process(real, imag, max_iters);
    }
	}
}

void saxpy (int nblocks, int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, double *device_value) {

  kernel_saxpy<<<nblocks, nthreads>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);
}
