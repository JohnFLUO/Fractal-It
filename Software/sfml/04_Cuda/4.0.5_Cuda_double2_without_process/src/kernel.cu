#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Color/ColorSmooth/ColorSmooth.hpp"
#include <stdio.h>

#include <hip/hip_vector_types.h>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <math_functions.h>

__global__ void kernel_compute(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;
  double2 z1;
  double2 z2;

  if ( (i<IMAGE_WIDTH) && (j<IMAGE_HEIGHT) ) {
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (j * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+ (i * zoom);

    z1.x = z.x;
    z1.y = z.y;

    device_value[j*IMAGE_WIDTH + i] = max_iters - 1;

    for (unsigned short counter = 0; counter < max_iters; counter++) {
      z2.x = z1.x * z1.x;
      z2.y = z1.y * z1.y;

      z1.y = 2.0f * z1.x * z1.y + z.y;
      z1.x = z2.x - z2.y + z.x;
      if ( (z2.x + z2.y) > 4.0f) {
        device_value[j*IMAGE_WIDTH + i] = counter;
        break;
      }
    }

	}
}



void compute (int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}
