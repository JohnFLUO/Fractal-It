#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Color/ColorSmooth/ColorSmooth.hpp"
#include <stdio.h>
//#include "Convergence/double/Convergence_dp_x86.hpp"


__device__ unsigned short process(const float startReal, const float startImag, unsigned short max_iters)  {
  float zReal = startReal;
  float zImag = startImag;

  for (unsigned short counter = 0; counter < max_iters; counter++) {
    float r2 = zReal * zReal;
    float i2 = zImag * zImag;
    zImag = 2.0f * zReal * zImag + startImag;
    zReal = r2 - i2 + startReal;
    if ( (r2 + i2) > 4.0f) {
      return counter;
    }
  }
  return max_iters - 1;
}


__global__ void kernel_compute(float zoom, float offsetX, float offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  float imag;
  float real;

  if ( (x<IMAGE_WIDTH) && (y<IMAGE_HEIGHT) ) {
    imag = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (y * zoom);
    real = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+((x)*zoom);
    device_value[y*IMAGE_WIDTH +x] = process(real, imag, max_iters);
	}
}

void compute (int nthreads, float zoom, float offsetX, float offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}
