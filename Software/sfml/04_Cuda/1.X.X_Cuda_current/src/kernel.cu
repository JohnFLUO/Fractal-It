#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Color/ColorSmooth/ColorSmooth.hpp"
#include <stdio.h>

#include <hip/hip_vector_types.h>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <math_functions.h>


__global__ void kernel_compute(double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double2 z;
  double2 z1;
  double2 z2;

  double device_value_copy;

  if ( (i<IMAGE_WIDTH) && (j<IMAGE_HEIGHT) ) {
    z.y = __fma_rn(j, zoom, __dsub_rn(offsetY, __dmul_rn(__ddiv_rn(IMAGE_HEIGHT, 2.0f), zoom)));
    z.x = __fma_rn(i, zoom, __dsub_rn(offsetX, __dmul_rn(__ddiv_rn(IMAGE_WIDTH,  2.0f), zoom)));

    /*
    z.y = offsetY - IMAGE_HEIGHT / 2.0f * zoom + (j * zoom);
    z.x = (offsetX - IMAGE_WIDTH / 2.0f * zoom)+ (i * zoom);
    */
    z1 = z;

    device_value_copy = __usad(max_iters, 1, 0);
    //device_value[j*IMAGE_WIDTH + i] = max_iters - 1;

    for (unsigned short counter = 0; counter < max_iters; counter++) {
      z2.x = __dmul_rn( z1.x, z1.x);
      z2.y = __dmul_rn( z1.y, z1.y);

      z1.y = __fma_rn(__dmul_rn(2.0f, z1.x), z1.y, z.y);
      z1.x = __dadd_rn(__dsub_rn(z2.x, z2.y), z.x);

/*
      z2.x = z1.x * z1.x;
      z2.y = z1.y * z1.y;

      z1.y = 2.0f * z1.x * z1.y + z.y;
      z1.x = z2.x - z2.y + z.x;
*/
      if ( __dadd_rn(z2.x, z2.y) > 4.0f) {
        device_value_copy = counter;
        //device_value[j*IMAGE_WIDTH + i] = counter;
        break;
      }
    }


    device_value[j*IMAGE_WIDTH + i] = device_value_copy;
	}
}



void compute (int nthreads, double zoom, double offsetX, double offsetY, unsigned int max_iters,
  int IMAGE_WIDTH, int IMAGE_HEIGHT, unsigned short *device_value) {

  int numBlocksX = ( IMAGE_WIDTH + ( nthreads - 1 ) ) / nthreads;
  int numBlocksY = ( IMAGE_HEIGHT + ( nthreads - 1 ) ) / nthreads;

  dim3 DimBlock(nthreads, nthreads, 1);
  dim3 DimGrid(numBlocksX, numBlocksY);

  kernel_compute<<<DimGrid, DimBlock>>>( zoom, offsetX, offsetY, max_iters,
    IMAGE_WIDTH, IMAGE_HEIGHT, device_value);

}
